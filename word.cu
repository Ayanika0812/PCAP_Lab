#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define WORD_LEN 10
#define TOTAL_WORDS 4

__constant__ char d_searchWord[WORD_LEN];

// Kernel to search for the word
__global__ void searchWordKernel(char *d_words, int *result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < TOTAL_WORDS) {
        char word[WORD_LEN];
        for (int i = 0; i < WORD_LEN; i++) {
            word[i] = d_words[idx * WORD_LEN + i];
        }

        // Compare word with search word in constant memory
        bool match = true;
        for (int i = 0; i < WORD_LEN; i++) {
            if (word[i] != d_searchWord[i]) {
                match = false;
                break;
            }
        }

        if (match) {
            *result = idx;
        }
    }
}

int main() {
    const char h_words[TOTAL_WORDS][WORD_LEN] = {"Apple", "Banana", "Mango", "Grape"};
    const char h_searchWord[WORD_LEN] = "Mango";

    char *d_words;
    int *d_result, h_result = -1;

    size_t size = TOTAL_WORDS * WORD_LEN * sizeof(char);
    hipMalloc((void**)&d_words, size);
    hipMemcpy(d_words, h_words, size, hipMemcpyHostToDevice);

    // Copy search word to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_searchWord), h_searchWord, WORD_LEN * sizeof(char));

    hipMalloc((void**)&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Kernel launch with 1D block and 1D grid
    searchWordKernel<<<1, 8>>>(d_words, d_result);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    if (h_result != -1) {
        printf("Word found at index %d\n", h_result);
    } else {
        printf("Word not found\n");
    }

    printf("Time taken: %f ms\n", milliseconds);

    // Free memory
    hipFree(d_words);
    hipFree(d_result);

    return 0;
}

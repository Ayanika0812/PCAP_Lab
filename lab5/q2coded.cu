
#include <hip/hip_runtime.h>
#include<stdio.h>
// #include "cuda_runtime.h"
// #include "device_launch_parameters.h"

__device__ int getGTID(){
    int blockid = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadid = blockid * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadid;
}

__global__ void add(int *a, int *b, int *c, int *n){
    int gtid = getGTID();
    if (gtid < *n){
        c[gtid] = a[gtid] + b[gtid];
    }
}

int main(){
    int *a, *b, *c;
    int n;   
    printf("Enter the size of the vectors: ");
    scanf("%d", &n);

    int s = n * sizeof(int);

    a = (int *)malloc(s);
    b = (int *)malloc(s);
    c = (int *)malloc(s);

    // Hardcode values for arrays a and b
    for (int i = 0; i < n; i++) {
        a[i] = i + 1;  // Array A: 1, 2, 3, ..., n
        b[i] = (i + 1) * 2;  // Array B: 2, 4, 6, ..., 2n
    }

    int *d_a, *d_b, *d_c, *d_n;
    hipMalloc((void **)&d_a, s);
    hipMalloc((void **)&d_b, s);
    hipMalloc((void **)&d_c, s);
    hipMalloc((void **)&d_n, sizeof(int));  // treat it as a pointer only
    
    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, s, hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    add<<<ceil(n / 256.0), 256>>>(d_a, d_b, d_c, d_n);  // needs to be float for ceil
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);

    // Display the unused threads for efficiency
    printf("Unused : %.0f   \n", (ceil(n / 256.0) * 256) - n);

    // Display the arrays a, b, and c
    printf("Array A: ");
    for (int i = 0; i < n; i++) {
        printf("%d  ", a[i]);
    }
    printf("\n");

    printf("Array B: ");
    for (int i = 0; i < n; i++) {
        printf("%d  ", b[i]);
    }
    printf("\n");

    // Display the result of C array
    printf("Result C (A + B): ");
    for (int i = 0; i < n; i++) {
        printf("%d  ", c[i]);
    }
    printf("\n");

    // Free the GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("Reg no. 220905128\n");  
}

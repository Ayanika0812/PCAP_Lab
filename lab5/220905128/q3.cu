
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void sine_angles(double *a, double *b, int *n){
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gtid < *n){
		b[gtid] = sin(a[gtid]);
	}
}

int main(){
	double *a, *b;
	int n;
	printf("Enter the size of the array: ");
	scanf("%d", &n);
	int s = n*sizeof(double);

	a = (double *)malloc(s);
	b = (double *)malloc(s);

	printf("Enter angles in rad: ");
	for (int i=0;i<n;i++){
		scanf("%lf", &a[i]);
	}

	double *d_a, *d_b;
	int *d_n;
	hipMalloc((void **)&d_a, s);
	hipMalloc((void **)&d_b, s);
	hipMalloc((void **)&d_n, sizeof(int));
	
	hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	sine_angles<<<ceil(n/256.0), 256>>>(d_a, d_b, d_n);
	hipMemcpy(b, d_b, s, hipMemcpyDeviceToHost);
   printf("Unused : %.0f   \n",(ceil(n/256.0)*256)-n);
	printf("Result : \n");
	for (int i=0;i<n;i++){
		printf("sine(%lf) -> %lf  \n", a[i], b[i]);
	}
	printf("\n");
	hipFree(d_a);
	hipFree(d_b);
    printf("Reg no. 220905128\n");	
}


/*
Enter the size of the array: 4
Enter angles in rad: 2.14 4.3 8.9 3.14
Unused : 252   
Result : 
sine(2.140000) -> 0.842330  
sine(4.300000) -> -0.916166  
sine(8.900000) -> 0.501021  
sine(3.140000) -> 0.001593  

Reg no. 220905128

*/
/*Write a program in CUDA which displays a shopping mall menu with its price. The N number of friends are allowed to purchase as many items they want. Calculate the totsl purchase done by N friends
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX_ITEMS 10
#define MAX_FRIENDS 100

// Structure to store item price
typedef struct {
    float price;
} Item;

// Kernel to calculate the total purchase for each friend
__global__ void calculatePurchase(float* purchases, int* itemIndices, const float* itemPrices, int numFriends, int numItems) {
    int friendId = blockIdx.x * blockDim.x + threadIdx.x;

    if (friendId < numFriends) {
        float total = 0.0f;

        // Loop through the items each friend buys
        for (int i = 0; i < numItems; i++) {
            int itemId = itemIndices[friendId * numItems + i];
            if (itemId >= 0 && itemId < numItems) {  // Check if itemId is valid
                total += itemPrices[itemId];
            }
        }

        purchases[friendId] = total;
    }
}

int main() {
    // Hardcoded items and prices
    Item items[MAX_ITEMS] = {
        {25.0f},  // Shirt
        {50.0f},  // Shoes
        {40.0f},  // Jeans
        {15.0f},  // Hat
        {30.0f},  // Bag
        {60.0f},  // Jacket
        {20.0f},  // T-shirt
        {5.0f},   // Socks
        {10.0f},  // Gloves
        {12.0f}   // Scarf
    };

    const int numItems = 10;  // Hardcoded number of items
    int numFriends;

    // Get the number of friends
    printf("Enter the number of friends: ");
    scanf("%d", &numFriends);

    // Ensure the number of friends doesn't exceed the maximum limit
    if (numFriends > MAX_FRIENDS) {
        printf("The number of friends can't exceed %d.\n", MAX_FRIENDS);
        return -1;
    }

    // Display the shopping mall menu
    printf("\nShopping Mall Menu:\n");
    for (int i = 0; i < numItems; i++) {
        printf("%d. Item %d - $%.2f\n", i + 1, i + 1, items[i].price);
    }

    // Get the purchases from each friend (which items they want to buy)
    int* itemIndices = (int*)malloc(numFriends * numItems * sizeof(int)); // Array for item selections

    // Each friend will purchase a number of items (let's say up to numItems for simplicity)
    for (int i = 0; i < numFriends; i++) {
        printf("\nEnter the items bought by Friend %d (enter item numbers, 0 to stop):\n", i + 1);
        
        // Initialize each friend's purchase array to -1 (invalid index)
        for (int j = 0; j < numItems; j++) {
            itemIndices[i * numItems + j] = -1;
        }

        // User input for item choices
        for (int j = 0; j < numItems; j++) {
            printf("Item number to purchase (1 to %d, 0 to stop): ", numItems);
            int itemChoice;
            scanf("%d", &itemChoice);

            if (itemChoice == 0) break; // Stop if 0 is entered

            // Store the item choice (convert to 0-based index)
            itemIndices[i * numItems + j] = itemChoice - 1;
        }
    }

    // Allocate memory on the device
    float* d_purchases;
    int* d_itemIndices;
    float* d_itemPrices;

    hipMalloc((void**)&d_purchases, numFriends * sizeof(float));
    hipMalloc((void**)&d_itemIndices, numFriends * numItems * sizeof(int));
    hipMalloc((void**)&d_itemPrices, numItems * sizeof(float));

    // Initialize purchases on the device to 0 before computation
    hipMemset(d_purchases, 0, numFriends * sizeof(float));

    // Copy data to the device
    hipMemcpy(d_itemIndices, itemIndices, numFriends * numItems * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_itemPrices, &items[0].price, numItems * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel to calculate total purchases for each friend
    int blockSize = 256;
    int numBlocks = (numFriends + blockSize - 1) / blockSize;
    calculatePurchase<<<numBlocks, blockSize>>>(d_purchases, d_itemIndices, d_itemPrices, numFriends, numItems);

    // Allocate memory for the results and copy back
    float* purchases = (float*)malloc(numFriends * sizeof(float));
    hipMemcpy(purchases, d_purchases, numFriends * sizeof(float), hipMemcpyDeviceToHost);

    // Display the total purchase for each friend
    printf("\nTotal purchases made by each friend:\n");
    for (int i = 0; i < numFriends; i++) {
        printf("Friend %d: $%.2f\n", i + 1, purchases[i]);
    }

    // Cleanup
    hipFree(d_purchases);
    hipFree(d_itemIndices);
    hipFree(d_itemPrices);

    free(itemIndices);
    free(purchases);

    return 0;
}


/*

student@lpcp-19:~/220905128/week11$ nvcc ad1.cu -o ad1
student@lpcp-19:~/220905128/week11$ ./ad1
Enter the number of friends: 2

Shopping Mall Menu:
1. Item 1 - $25.00
2. Item 2 - $50.00
3. Item 3 - $40.00
4. Item 4 - $15.00
5. Item 5 - $30.00
6. Item 6 - $60.00
7. Item 7 - $20.00
8. Item 8 - $5.00
9. Item 9 - $10.00
10. Item 10 - $12.00

Enter the items bought by Friend 1 (enter item numbers, 0 to stop):
Item number to purchase (1 to 10, 0 to stop): 2
Item number to purchase (1 to 10, 0 to stop): 3
Item number to purchase (1 to 10, 0 to stop): 0

Enter the items bought by Friend 2 (enter item numbers, 0 to stop):
Item number to purchase (1 to 10, 0 to stop): 2
Item number to purchase (1 to 10, 0 to stop): 4
Item number to purchase (1 to 10, 0 to stop): 5
Item number to purchase (1 to 10, 0 to stop): 0

Total purchases made by each friend:
Friend 1: $90.00
Friend 2: $95.00

*/
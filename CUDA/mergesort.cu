#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void merge(int *arr, int left, int mid, int right) {
    int n1 = mid - left + 1;
    int n2 = right - mid;

    int *L = new int[n1];
    int *R = new int[n2];

    for (int i = 0; i < n1; i++) L[i] = arr[left + i];
    for (int i = 0; i < n2; i++) R[i] = arr[mid + 1 + i];

    int i = 0, j = 0, k = left;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            arr[k] = L[i];
            i++;
        } else {
            arr[k] = R[j];
            j++;
        }
        k++;
    }

    while (i < n1) {
        arr[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        arr[k] = R[j];
        j++;
        k++;
    }

    delete[] L;
    delete[] R;
}

// Kernel function to perform merge sort
__global__ void mergeSortKernel(int *arr, int left, int right) {
    if (left < right) {
        int mid = left + (right - left) / 2;

        // Sort first half
        mergeSortKernel<<<1, 1>>>(arr, left, mid);

        // Sort second half
        mergeSortKernel<<<1, 1>>>(arr, mid + 1, right);

        // Merge the sorted halves
        merge(arr, left, mid, right);
    }
}

// Host function to initiate the merge sort
void mergeSort(int *arr, int n) {
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to perform merge sort
    mergeSortKernel<<<1, 1>>>(d_arr, 0, n - 1);

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_arr);
}

int main() {
    int arr[] = {3, 2, 5, 1, 4};
    int n = sizeof(arr) / sizeof(arr[0]);

    mergeSort(arr, n);

    printf("Sorted array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");

    return 0;
}
